#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <cstddef>
#include <cstdio>
#define BLOCK_SIZE 512

// Define your kernels in this file you may use more than one kernel if you
// need to

__global__ void preScanKernel(float *inout, unsigned size, float *sum) {
  __shared__ float temp[BLOCK_SIZE * 2];

  int t = threadIdx.x;
  int n = size;
  int offset = 1;

  // load input into shared memory
  temp[2 * t] = inout[2 * t];
  temp[2 * t + 1] = inout[2 * t + 1];

  __syncthreads();

  // Up-Sweep
  for (int d = n >> 1; d > 0; d >>= 1) {
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }

  // clear the last element
  if (t == 0) {
    temp[n - 1] = 0;
  }

  __syncthreads();

  // traverse down tree & build scan
  for (int d = 1; d < n; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (t < d) {
      int ai = offset * (2 * t + 1) - 1;
      int bi = offset * (2 * t + 2) - 1;
      float val = temp[ai];
      if (bi < BLOCK_SIZE * 2 && ai < BLOCK_SIZE * 2) {
        temp[ai] = temp[bi];
      }
      if (bi < BLOCK_SIZE * 2) {
        temp[bi] += val;
      }
    }
  }

  __syncthreads();

  inout[2 * t] = temp[2 * t];
  inout[2 * t + 1] = temp[2 * t + 1];
}

__global__ void addKernel(float *inout, float *sum, unsigned size) {
  unsigned int t = threadIdx.x;
  unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;
  unsigned int globalIndex = start + t;

  if (blockIdx.x > 0) {
    if (globalIndex < size) {
      inout[globalIndex] += sum[blockIdx.x - 1];
    }
    if (globalIndex + BLOCK_SIZE < size) {
      inout[globalIndex + BLOCK_SIZE] += sum[blockIdx.x - 1];
    }
  }
}

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void preScan(float *inout, unsigned in_size) {
  float *sum;
  float *paddedInout;
  unsigned num_blocks;
  hipError_t cuda_ret;
  dim3 dim_grid, dim_block;

  num_blocks = in_size / (BLOCK_SIZE * 2);
  if (in_size % (BLOCK_SIZE * 2) != 0)
    num_blocks++;

  dim_block.x = BLOCK_SIZE;
  dim_block.y = 1;
  dim_block.z = 1;
  dim_grid.x = num_blocks;
  dim_grid.y = 1;
  dim_grid.z = 1;

  unsigned paddedSize = nextPow2(in_size);

  float *real_inout = inout;
  unsigned real_size = in_size;

  // Check if padding is needed
  if (paddedSize != in_size) {
    printf("Padding needed\n");

    cuda_ret = hipMalloc((void **)&paddedInout, paddedSize * sizeof(float));
    if (cuda_ret != hipSuccess)
      FATAL("Unable to allocate device memory for padded inout");

    cuda_ret = hipMemcpy(paddedInout, inout, in_size * sizeof(float),
                          hipMemcpyHostToDevice);
    if (cuda_ret != hipSuccess)
      FATAL("Unable to copy to padded inout");

    // Initialize the padded elements to 0
    cuda_ret = hipMemset(paddedInout + in_size, 0,
                          (paddedSize - in_size) * sizeof(float));
    if (cuda_ret != hipSuccess)
      FATAL("Unable to initialize padded elements");

    real_inout = paddedInout; // Use the padded array
    real_size = paddedSize;
  }

  // print the padded size
  printf("Using size: %d\n", in_size);

  if (num_blocks > 1) {
    cuda_ret = hipMalloc((void **)&sum, num_blocks * sizeof(float));
    if (cuda_ret != hipSuccess)
      FATAL("Unable to allocate device memory");

    preScanKernel<<<dim_grid, dim_block>>>(inout, real_size, sum);
    // preScan(sum, num_blocks);
    // addKernel<<<dim_grid, dim_block>>>(inout, sum, in_size);

    hipFree(sum);
  } else {
    preScanKernel<<<dim_grid, dim_block>>>(real_inout, real_size, NULL);
  }

  if (paddedSize != in_size) {
    printf("Copying back\n");
    // copy data from padded input to original input
    cuda_ret = hipMemcpy(inout, paddedInout, in_size * sizeof(float),
                          hipMemcpyDeviceToDevice);

    if (cuda_ret != hipSuccess) {
      fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(cuda_ret));
      FATAL("Unable to copy from padded inout");
    }

    cuda_ret = hipFree(paddedInout);
    if (cuda_ret != hipSuccess)
      FATAL("Unable to free padded inout");
  }
}
