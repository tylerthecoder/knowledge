#include <stdio.h>
#include <iostream>
#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    time_t t;

    /* Intializes random number generator */
    srand((unsigned) time(&t));

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    hipError_t cuda_ret;
    int  height, width;
    dim3 dim_grid, dim_block;

	/* Read image dimensions */
    if (argc == 1) {
        height = 1400;
	    width =1400;
    } else if (argc == 2) {
        height= atoi(argv[1]);
	width= atoi(argv[1]);
    } else if (argc == 3) {
        height = atoi(argv[1]);
        width = atoi(argv[2]);
    }else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./gameoflife          # Matrix is 1400 x 1400"
           "\n    Usage: ./gameoflife <m>      # Matrix is m x m"
	   "\n    Usage: ./gameoflife <m> <n>  # Matrix is m x n"
           "\n");
        exit(0);
    }



	/* Allocate host memory */
	int *grid=new int [height*width*2];
	int *Ggrid_result=new int [height*width*2];
	/* Initialize Matrix */
	InitialGrid(grid,height,width);
	GiveLife(0,height*width/2,grid,height,width);


    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("\nThe size of the universe is %d x %d.\n\n", height, width);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);
    int *GPUgrid;

    long long int size=sizeof(int)*2*width*height;
    cuda_ret = (hipMalloc((void**) &GPUgrid, size));
    if(cuda_ret != hipSuccess) {
		printf("Unable to allocate GPU global memory");
		exit(-1);
	}

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret =(hipMemcpy(GPUgrid,grid,size,hipMemcpyHostToDevice));
    if(cuda_ret != hipSuccess) {
		printf("Unable to copy to GPU memory");
		exit(-1);
	}

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    // Just before launching the kernel
    dim_block = dim3(16, 16);  // For example, blocks of 16x16 threads
    dim_grid = dim3((width + dim_block.x - 1)/dim_block.x, (height + dim_block.y - 1)/dim_block.y);

    // INSERT CODE ABOVE
	int select =0;
	for(int m=0;m<ITERATION;m++){
        	GameofLife<<<dim_grid, dim_block>>>(GPUgrid,select,width,height);
         	select=1-select;
        }
	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) {
		printf("Unable to launch/execute kernel");
		exit(-1);
	}

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host...\n"); fflush(stdout);
    startTime(&timer);

    hipMemcpy(Ggrid_result,GPUgrid,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

   //CPU -----------------------------------------------------------------------

	int nowGrid=0;
        for(int n=0;n<ITERATION;n++)
        {
		GameofLife_CPU( grid, width, height,nowGrid);
		nowGrid=1-nowGrid;

      	}

// Verify correctness -----------------------------------------------------
	printf("Verifying..."); fflush(stdout);
	verify(Ggrid_result,grid,height,width);

// Free memory ------------------------------------------------------------

    cuda_ret = hipFree(GPUgrid);
    if(cuda_ret != hipSuccess) {
		printf("Unable free cuda memory");
		exit(-1);
	}

	 delete [] grid;
	 delete [] Ggrid_result;
	 return 0;
}
