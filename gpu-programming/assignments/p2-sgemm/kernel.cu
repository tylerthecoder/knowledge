#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include </home/tylord/assignments/gpu-programming/cuda-headers/hip/hip_runtime.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    // Calculate row index of the C matrix and column index of the C matrix

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    if (row < m && col < n) {
        // Traverse each column of A and each row of B to compute the matrix multiplication
        for (int i = 0; i < k; ++i) {
            sum += A[row * k + i] * B[i * n + col];
        }
        // Write the computed value to the C matrix
        C[row * n + col] = sum;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE
    dim3 dim_grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);


    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);



}


