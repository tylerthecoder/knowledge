/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    time_t t;


    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    /* Intializes random number generator */
    srand((unsigned) time(&t));

    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    float* A_d;
    float* B_d;
    float* C_d;

    // Allocate device memory
    hipMalloc((void**)&A_d, sizeof(float)*n);
    hipMalloc((void**)&B_d, sizeof(float)*n);
    hipMalloc((void**)&C_d, sizeof(float)*n);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    // Copy data from host to device
    hipMemcpy(A_d, A_h, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, sizeof(float)*n, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    // Defining block size and calculating grid size
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);

    // Launching the kernel
    vecAddKernel<<<gridSize, blockSize>>>(A_d, B_d, C_d, n);

    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) {
		printf("Unable to launch kernel\n");
		exit(-1);
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    hipMemcpy(C_h, C_d, sizeof(float)*n, hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);


    return 0;

}

